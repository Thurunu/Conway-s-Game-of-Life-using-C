#include <stdio.h>
#include <stdlib.h>
#include <unistd.h>
#include <hip/hip_runtime.h>

// Define grid dimensions and game speed
#define WIDTH 190
#define HEIGHT 50

#define SPEED 50 // Speed in milliseconds for the next generation

// Define characters used for display
#define BACKGROUND '.'
#define CELL '0'

// Define the possible states of a cell (DEAD or ALIVE)
typedef enum
{
    DEAD,
    ALIVE
} State;

// Structure representing a cell, containing its state and symbol
typedef struct
{
    State state;
    char symbol;
} Cell;

// Host 2D array representing the grid of cells
Cell h_grid[HEIGHT][WIDTH] = { { {DEAD, BACKGROUND} } };

// CUDA Kernel to generate the next state of the grid based on Game of Life rules
__global__ void gen_next(Cell *d_grid, Cell *d_next_grid)
{
    int i = blockIdx.y * blockDim.y + threadIdx.y;
    int j = blockIdx.x * blockDim.x + threadIdx.x;

    if (i < HEIGHT && j < WIDTH)
    {
        int alive_count = 0; // Counter to track number of alive neighbors

        // Loop through the neighbors (3x3 grid around the current cell)
        for (int k = -1; k <= 1; k++)
        {
            for (int l = -1; l <= 1; l++)
            {
                // Skip the current cell itself
                if (k == 0 && l == 0)
                    continue;

                int row = (i + k + HEIGHT) % HEIGHT;
                int col = (j + l + WIDTH) % WIDTH;

                if (d_grid[row * WIDTH + col].state == ALIVE)
                    alive_count++;
            }
        }

        // Apply the rules of the Game of Life based on the count of alive neighbors
        if (d_grid[i * WIDTH + j].state == ALIVE)
        {
            if (alive_count < 2 || alive_count > 3)
            {
                d_next_grid[i * WIDTH + j].state = DEAD;  // Underpopulation or overpopulation
            }
            else
            {
                d_next_grid[i * WIDTH + j].state = ALIVE; // Lives to next generation
            }
        }
        else
        {
            if (alive_count == 3)
            {
                d_next_grid[i * WIDTH + j].state = ALIVE; // Reproduction
            }
            else
            {
                d_next_grid[i * WIDTH + j].state = DEAD;
            }
        }
    }
}

// Function to initialize the grid, setting all cells to DEAD
void init_grid()
{
    for (size_t i = 0; i < HEIGHT; i++)
    {
        for (size_t j = 0; j < WIDTH; j++)
        {
            h_grid[i][j].state = DEAD; // Initialize all cells as DEAD
        }
    }
}

// Function to print the grid to the console
int print_grid()
{
    int alive_count = 0; // Counter to track the number of alive cells
    for (size_t i = 0; i < HEIGHT; i++)
    {
        for (size_t j = 0; j < WIDTH; j++)
        {
            // Print the CELL symbol if the cell is alive, otherwise print the BACKGROUND
            if (h_grid[i][j].state == ALIVE)
            {
                alive_count++;
                printf("%c", CELL);
            }
            else
            {
                printf("%c", BACKGROUND);
            }
        }
        printf("\n"); // New line after each row
    }
    return alive_count; // Return the total number of alive cells
}

// Main function
int main()
{
    init_grid(); // Initialize the grid with all cells set to DEAD

    // Set up an initial configuration of live cells (a small block of cells is made ALIVE)
    for (size_t i = 0; i < WIDTH / 5; i++)
    {
        for (size_t j = 0; j < HEIGHT / 5; j++)
        {
            h_grid[j][i].state = ALIVE; // Set some cells in the top-left corner to ALIVE
        }
    }

    // Allocate memory on the device (GPU)
    Cell *d_grid, *d_next_grid;
    hipMalloc((void **)&d_grid, WIDTH * HEIGHT * sizeof(Cell));
    hipMalloc((void **)&d_next_grid, WIDTH * HEIGHT * sizeof(Cell));

    // Define the number of threads and blocks
    dim3 threadsPerBlock(16, 16);
    dim3 blocksPerGrid((WIDTH + threadsPerBlock.x - 1) / threadsPerBlock.x, (HEIGHT + threadsPerBlock.y - 1) / threadsPerBlock.y);

    system("clear");
    // Loop to run the simulation continuously until no more live cells exist
    while (print_grid() != 0)
    {
        // Copy host grid to device
        hipMemcpy(d_grid, h_grid, WIDTH * HEIGHT * sizeof(Cell), hipMemcpyHostToDevice);

        // Launch the kernel to compute the next generation on the GPU
        gen_next<<<blocksPerGrid, threadsPerBlock>>>(d_grid, d_next_grid);

        // Swap grids: d_next_grid will become the new d_grid in the next iteration
        Cell *temp = d_grid;
        d_grid = d_next_grid;
        d_next_grid = temp;

        // Copy the updated grid back to the host
        hipMemcpy(h_grid, d_grid, WIDTH * HEIGHT * sizeof(Cell), hipMemcpyDeviceToHost);

        usleep(SPEED * 1000); // Pause the program for a short period (to control the speed of the simulation)
        system("clear");      // Clear the console to show the updated grid
    }

    // Free device memory
    hipFree(d_grid);
    hipFree(d_next_grid);

    return 0;
}
